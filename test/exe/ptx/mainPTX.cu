#include "hip/hip_runtime.h"
/**
 * @file main.cu
 *
 * @brief main CUDA file of the project. To be compiled into PTX.
 *
 * @version 0.1
 * @date 2022-09-03
 */
#include <vector>
#include <cstddef>
#include <hip/hip_runtime.h>

#include <thrust/device_vector.h>

namespace
{
  std::size_t constexpr N = 10;

  __global__ void my_kernel(float *const in)
  {
    auto const i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i >= N)
    {
      return;
    }

    in[i] = 1.0f;
  }
}

void cuda_kernel_ptx()
{
  std::cout << "Hello from the PTX file of the executable" << std::endl;
  std::vector<float> inHost(N, 1.0f);
  thrust::device_vector<float> inGPU(inHost);

  my_kernel<<<1, N>>>(thrust::raw_pointer_cast(inGPU.data()));
}
